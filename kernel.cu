﻿#include <iostream>
#include <sstream>
#include <stdlib.h>
#include "hip/hip_runtime.h"


using namespace std;


/* This is our CUDA call wrapper, we will use in PAC.
*
*  Almost all CUDA calls should be wrapped with this makro.
*  Errors from these calls will be catched and printed on the console.
*  If an error appears, the program will terminate.
*
* Example: gpuErrCheck(cudaMalloc(&deviceA, N * sizeof(int)));
*          gpuErrCheck(cudaMemcpy(deviceA, hostA, N * sizeof(int), cudaMemcpyHostToDevice));
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
		if (abort)
		{
			exit(code);
		}
	}
}


// Compare result vectors
int compareResultVec(int* vectorCPU, int* vectorGPU, int size)
{
	int error = 0;
	for (int i = 0; i < size; i++)
	{
		error += abs(vectorCPU[i] - vectorGPU[i]);
	}
	if (error == 0)
	{
		cout << "No errors. All good!" << endl;
		return 0;
	}
	else
	{
		cout << "Accumulated error: " << error << endl;
		return -1;
	}
}


// Very inefficient way to check if a number is prime
// Use Baillie–PSW primality test or ECPP if you want to do it right :)
bool isPrime(int n)
{
	if (n == 2 || n == 3)
		return true;

	if (n <= 1 || n % 2 == 0 || n % 3 == 0)
		return false;

	for (int i = 5; i * i <= n; i += 6)
	{
		if (n % i == 0 || n % (i + 2) == 0)
			return false;
	}

	return true;
}


//CPU implementation of compact pattern, returns number of found prime values.
int compact_prime(int* input, int* output, int size)
{
	int current_pos = 0;
	for (int i = 0; i < size; ++i)
	{
		int foo = input[i];
		if (isPrime(foo)) {
			output[current_pos] = input[i];
			current_pos += 1;
		}
	}
	return current_pos;
}


int main(void)
{
	// Define the size of the vector
	const int size = 1 << 22;
	// This gives you 2048 * 2048 items,
	// sounds like a perfect 2 stage fit for the sum scan implementation

	// Allocate and prepare input vector
	int* hostVector = new int[size];
	srand(1337);  // We have the same pseudo-random numbers each time
	for (int index = 0; index < size; ++index) {
		hostVector[index] = rand();
	}

	// Make things easy, so use a same sized output buffer
	int* hostOutput_CPU = new int[size];
	int found_primes = compact_prime(hostVector, hostOutput_CPU, size);
	cout << "Found " << found_primes << "prime numbers." << endl;

	// ToDo: Implement compact pattern on GPU, you can use var found_primes
	// to allocate the right size or to only loop/check what is needed in 
	// comparing the results (in case of an in-place implementation).

	// Free memory on device & host
	delete[] hostVector;
	delete[] hostOutput_CPU;

	return 0;
}